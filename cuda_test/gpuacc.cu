#include "hip/hip_runtime.h"
//#include "gpuacc.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <hipfft/hipfft.h>
#include "hipblas.h"
#include <stdio.h>
#include <stdlib.h>

inline void GPUassert(hipError_t code, char * file, int line, bool Abort = true)
{
	if (code != 0) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (Abort) exit(code);
	}
}

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }

__host__ __device__ void swap(char *x, char *y)
{
	char temp;
	temp = *x;
	*x = *y;
	*y = temp;
}

__device__ void permute_device(char *a, int i, int n, int tid, int* count)
{
	if (i == n) {
		char b[4]; char* c = a - 1;
		b[0] = c[0]; b[1] = c[1]; b[2] = c[2]; b[3] = c[3];
		printf("Permutation nr. %i from thread nr. %i is equal to %s\n", count[0], tid, c); count[0] = count[0] + 1;
	}
	else
	{
		for (int j = i; j <= n; j++)
		{
			swap((a + i), (a + j));
			permute_device(a, i + 1, n, tid, count);
			swap((a + i), (a + j)); //backtrack
		}
	}
}

__global__ void permute_kernel(char* d_A, int size) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int count[1]; count[0] = 0;

	char local_array[4];

	for (int i = 0; i < size; i++) {
		local_array[i] = d_A[i];
	}

	swap(local_array + threadIdx.x, local_array);
	permute_device(local_array + 1, 0, 2, tid, count);

}

void mygputest() {
	char h_a[] = "ABCD";
	char* d_a;
	hipMalloc((void**)&d_a, sizeof(h_a));
	GPUerrchk(hipMemcpy(d_a, h_a, sizeof(h_a), hipMemcpyHostToDevice));

	printf("\n\n Permutations on GPU\n");
	permute_kernel << <1, 4 >> > (d_a, 4);
	GPUerrchk(hipPeekAtLastError());
	GPUerrchk(hipDeviceSynchronize());

	getchar();

}