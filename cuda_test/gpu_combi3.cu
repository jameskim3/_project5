#include "hip/hip_runtime.h"
//#include "gpuacc.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <hipfft/hipfft.h>
#include "hipblas.h"
#include <stdio.h>
#include <stdlib.h>

inline void GPUassert(hipError_t code, char * file, int line, bool Abort = true)
{
	if (code != 0) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (Abort) exit(code);
	}
}

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }

__host__ __device__ void swap(char *x, char *y)
{
	char temp;
	temp = *x;
	*x = *y;
	*y = temp;
}

__device__ void combi_device(int s, int n, int cnt, int* sum, int tid)
{
	if (cnt == 10) {
		sum[0] = sum[0] + 1;
		return;
	}
	for (int i = s; i < n; i++) {
		combi_device(i + 1, n, cnt + 1, sum, tid);
	}
}

__global__ void combi_kernel(int* d_A, int size) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int sum[1]; sum[0] = 0;

	int* local_array = new int[4 * 1000];
	int* visited = new int[4 * 10];

	for (int i = 0; i < 4 * 1000; i++) {
		local_array[i] = 0;
	}
	for (int i = 0; i < 4 * 10; i++) {
		visited[i] = 0;
	}

	combi_device(0, 30, 0, sum, tid);
	printf("Combination count %i, thread nr. %i\n", sum[0], tid);

	delete[] local_array;
	delete[] visited;

}

//void mygputest() {
//	int h_a[1000] = { 0 };
//	int* d_a;
//	hipMalloc((void**)&d_a, sizeof(h_a));
//	GPUerrchk(hipMemcpy(d_a, h_a, sizeof(h_a), hipMemcpyHostToDevice));
//
//	printf("\n\n Combinations on GPU\n");
//	combi_kernel << <1, 1 >> > (d_a, 4);
//	GPUerrchk(hipPeekAtLastError());
//	GPUerrchk(hipDeviceSynchronize());
//
//}

const int N = 20;
const int K = 12;

const int SIZE = 70000;
const int CASE = 7000;

//__device__ int SUM[SIZE];
//__device__ int VAL[SIZE];
//__device__ int PRE[SIZE];
//__device__ int MAP[CASE*N*K];
//__device__ int CNT[10][N*K];
//__device__ int wp[10];


__device__ void combi(int tid, int n, int k, int* SUM, int* VAL, int*  PRE, int* MAP, int* CNT, int* wp) {
	wp[tid] = 0;
	for (int i = 0; i < N*K; i++)CNT[tid*N*K + i] = 0;
	int p;
	for (int i = 0; i <= n; i++) {
		for (int j = 0; j <= k && j <= i; j++) {
			int cnt = 0;
			if (j == 0) {
				PRE[wp[tid]] = 0;
				VAL[wp[tid]] = -1;
				SUM[wp[tid]] = 0;
				p = (i*K + j)*CASE;
				MAP[p + cnt++] = wp[tid]++;
				CNT[tid*N*K + i*K + j]++;
			}
			else if (j == i) {
				p = ((i - 1)*K + j - 1)*CASE;
				PRE[wp[tid]] = PRE[p];
				VAL[wp[tid]] = i;
				SUM[wp[tid]] = SUM[p] + i;
				p = (i*K + j)*CASE;
				MAP[p + cnt++] = wp[tid]++;
				CNT[tid*N*K + i*K + j]++;
			}
			else {
				int cnt = 0;
				int rep = 0;
				while (rep < CNT[tid*N*K + (i-1)*K + j-1]) {
					p = ((i - 1)*K + j - 1)*CASE;
					PRE[wp[tid]] = PRE[p + rep];
					VAL[wp[tid]] = i;
					SUM[wp[tid]] = SUM[p + rep++] + i;
					p = (i*K + j)*CASE;
					MAP[p + cnt++] = wp[tid]++;
					CNT[tid*N*K + i*K + j]++;
				}
				rep = 0;
				while (rep < CNT[tid*N*K + (i - 1)*K + j]) {
					p = ((i - 1)*K + j)*CASE;
					PRE[wp[tid]] = PRE[p + rep];
					VAL[wp[tid]] = -1;
					SUM[wp[tid]] = SUM[p + rep++];
					p = (i*K + j)*CASE;
					MAP[p + cnt++] = wp[tid]++;
					CNT[tid*N*K + i*K + j]++;
				}
			}
		}
	}
	//return CNT[n*K + k];
}

//void print_combi(int n, int k) {
//	int cnt = 100;//ret[n][k].cnt
//	for (int i = 0; i < cnt; i++) {
//		int rp = ret[n][k].arr[i];
//		CM* p = &map[rp];
//		while (p->val != -1) {
//			if (p->val != -2) {
//				printf("%2d,", p->val);
//			}
//			p = &map[p->pre];
//		}
//		printf("\n");
//	}
//}

__global__ void kernel_combi2(int start) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//int* SUM = new int[SIZE];
	//int* VAL = new int[SIZE];
	//int* PRE = new int[SIZE];
	//int* MAP = new int[CASE*N*K];
	//int* CNT = new int[10*N*K];
	//int* wp = new int[10];
	//int n = 15;
	//int k = 10;
	//for (int i = 0; i < 2; i++) {
	//	combi(tid, n, k, SUM, VAL, PRE, MAP, CNT, wp);
	//}
	//printf("Combination count %i, thread nr. %i\n", CNT[tid*N*K+n*K+k], tid);

	//delete[] SUM;
	//delete[] VAL;
	//delete[] PRE;
	//delete[] MAP;
	//delete[] CNT;
	//delete[] wp;

	for (int i = 0; i < 1000000; i++)
	{
		int a = 1;
	}
	printf("Combination count %i, thread nr. %i\n", tid, tid);
}

void mygputest() {
	//printf("%d_C_%d is %d\n", n, k, combi(n, k));
	//print_combi(n, k);

	for (int i = 0; i < 4000; i+=1000) {
		kernel_combi2 << <1, 10 >> > (i);
	}
}
