#include "hip/hip_runtime.h"
//#include "gpuacc.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <hipfft/hipfft.h>
#include "hipblas.h"
#include <stdio.h>
#include <stdlib.h>

inline void GPUassert(hipError_t code, char * file, int line, bool Abort = true)
{
	if (code != 0) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (Abort) exit(code);
	}
}

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }

__host__ __device__ void swap(char *x, char *y)
{
	char temp;
	temp = *x;
	*x = *y;
	*y = temp;
}

__device__ void combi_device2(int *cnt, int tid) {
	printf("Debug2\n"); 
	if (*cnt == 5)return;
//	*sum = *sum + idx;
	*cnt += 1;
	combi_device2(cnt, tid);
}

__device__ void combi_device(int *a, int *visited, int idx, int cnt, int *sum, int *rp, int tid)
{
	printf("Debug2\n");
	if (cnt == 5) {
		a[*rp] = *sum + tid;
		*rp += 1;
		printf("Combi nr. %i is %d\n", tid, *rp);
		return;
	}
	else
	{
		for (int i = idx; i < 10; i++)
		{
			if (visited[i] != 0) {
				visited[i] = 1;
				*sum += i;
				combi_device(a, visited, i+1, cnt+1, sum, rp, tid);
				visited[i] = 0;
			}
		}
	}
}

__global__ void permute_kernel(int* d_A, int size) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int count[1]; count[0] = 0;

	int local_visited[4][250] = { 0 };
	int local_sum[4] = { 0 };
	int rp[4] = { 0 };
	int local_array[4*1000];

	int cnt[1] = { 0 };

	printf("Debug\n");
	//combi_device(&local_array[tid * 1000], local_visited[tid], 0, 0, &local_sum[tid], &rp[tid], tid);
	combi_device2(cnt, tid);
	printf("Combi nr. %i is %d\n", tid, rp[tid]);
}

int h_a[4*1000] = { 0 };
void mygputest() {
	int* d_a;
	hipMalloc((void**)&d_a, sizeof(h_a));
	GPUerrchk(hipMemcpy(d_a, h_a, sizeof(h_a), hipMemcpyHostToDevice));

	printf("\n\n Combination on GPU\n");
	permute_kernel << <1, 4 >> > (d_a, 4);
	GPUerrchk(hipPeekAtLastError());
	GPUerrchk(hipDeviceSynchronize());


}