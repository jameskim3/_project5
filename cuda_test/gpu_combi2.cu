#include "hip/hip_runtime.h"
//#include "gpuacc.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <hipfft/hipfft.h>
#include "hipblas.h"
#include <stdio.h>
#include <stdlib.h>

inline void GPUassert(hipError_t code, char * file, int line, bool Abort = true)
{
	if (code != 0) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (Abort) exit(code);
	}
}

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }

__host__ __device__ void swap(char *x, char *y)
{
	char temp;
	temp = *x;
	*x = *y;
	*y = temp;
}

__device__ void combi_device(int s, int n, int cnt, int* sum, int tid)
{
	if (cnt ==10) {
		sum[0] = sum[0] + 1;
		return;
	}
	for (int i = s; i < n; i++)	{
		combi_device(i + 1, n, cnt+1, sum, tid);
	}
}

__global__ void combi_kernel(int* d_A, int size) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int sum[1]; sum[0] = 0;

	int* local_array = new int[4*1000];
	int* visited = new int[4 * 10];

	for (int i = 0; i < 4 * 1000; i++) {
		local_array[i] = 0;
	}
	for (int i = 0; i < 4 * 10; i++) {
		visited[i] = 0;
	}

	combi_device(0, 10, 0, sum, tid);
	printf("Combination count %i, thread nr. %i\n", sum[0], tid);
	
	delete[] local_array;
	delete[] visited;

}

void mygputest() {
	int h_a[1000] = { 0 };
	int* d_a;
	hipMalloc((void**)&d_a, sizeof(h_a));
	GPUerrchk(hipMemcpy(d_a, h_a, sizeof(h_a), hipMemcpyHostToDevice));

	printf("\n\n Combinations on GPU\n");
	combi_kernel << <1, 1 >> > (d_a, 4);
	GPUerrchk(hipPeekAtLastError());
	GPUerrchk(hipDeviceSynchronize());

}